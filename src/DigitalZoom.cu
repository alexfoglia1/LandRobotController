#include "hip/hip_runtime.h"
#include "DigitalZoom.h"
#include <opencv2/imgproc.hpp>

__global__ void zoomKernel(
    const uchar3* input, uchar3* output,
    int zoomWidth, int zoomHeight,
    int width, int height,
    size_t inputPitch, size_t outputPitch)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x > width || y > height) return;

    // Calcola la posizione originale
    float scaleX = (float)zoomWidth / width;
    float scaleY = (float)zoomHeight / height;
    int srcX = __min((int)(x * scaleX), zoomWidth);
    int srcY = __min((int)(y * scaleY), zoomHeight);

    // Calcola l'indice considerando il pitch (larghezza effettiva della riga in byte)
    uchar3* inputRow = (uchar3*)((char*)input + srcY * inputPitch);
    uchar3* outputRow = (uchar3*)((char*)output + y * outputPitch);

    outputRow[x] = inputRow[srcX];
}


void digitalZoom(cv::cuda::GpuMat& gpuMat, DigitalZoomStep zoomStep)
{
    dim3 gridSize(gpuMat.cols, gpuMat.rows);

    uchar3* output;
    size_t outputPitch;
    hipMallocPitch(&output, &outputPitch, gpuMat.cols * sizeof(uchar3), gpuMat.rows);

    float enlargeSize = powf(1.25f, static_cast<float>(zoomStep));
    dim3 blockSize(16, 16);

    int roiWidth = gpuMat.cols * (1.0f / (enlargeSize));
    int roiHeight = gpuMat.rows * (1.0f / (enlargeSize));
    int roiX = gpuMat.cols / 2 - roiWidth / 2;
    int roiY = gpuMat.rows / 2 - roiHeight / 2;

    cv::cuda::GpuMat inputRoi = gpuMat(cv::Rect(roiX, roiY, roiWidth, roiHeight));

    uchar3* input = inputRoi.ptr<uchar3>();
    zoomKernel << <gridSize, blockSize >> > (input, output,
        roiWidth, roiHeight,
        gpuMat.cols, gpuMat.rows,
        inputRoi.step, gpuMat.step);

    hipError_t err = hipMemcpy2D(gpuMat.ptr<uchar>(), gpuMat.step,
        output, outputPitch,
        gpuMat.cols * sizeof(uchar3), gpuMat.rows,
        hipMemcpyDeviceToDevice);
    
    if (err != hipSuccess)
    {
        std::cerr << "hipMemcpy failed: " << hipGetErrorString(err) << std::endl;
    }

    hipFree(output);

    hipDeviceSynchronize();
}